#include "hip/hip_runtime.h"
/*
* Author: Yicheng Zhang
* Association: UC Riverside
* Date: Sep 10th, 2023
*
* Description: 
* Description: Reverse engineering for nvlink packet/data flit
* Idea of program: send a small data packet from gpu A to gpu B. Use nvprof to profile nvlink transactions. 
* Candiate perf ctrs: nvlink_total_data_transmitted,nvlink_total_data_received,nvlink_overhead_data_transmitted,nvlink_overhead_data_received,nvlink_total_response_data_received,nvlink_user_response_data_received,nvlink_total_write_data_transmitted,nvlink_user_data_transmitted,nvlink_user_data_received,nvlink_user_write_data_transmitted
* Nvprof commands: nvprof  --profile-from-start off --devices 0 --aggregate-mode off --csv --log-file "file_name".csv --event-collection-mode continuous -m "ctr_name" ./nvlink_re size_to_transfer
*/

#include <vector>
#include <hip/hip_runtime_api.h> // For hipProfilerStart() and hipProfilerStop()
#include <cstdio>
#include <string>
#include <thrust/device_vector.h>
#include <fstream>
#include <cupti_profiler.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include "kernel.cu"
#include <iostream>
#include <chrono>
#include <thread>




 
int main(int argc, char **argv) {

    using namespace std;
    int local = 0; 
    int remote = 1;
    int sizeElement = 64;
    int *h_local, *h_remote;
    int *d_local, *d_remote;
    

    local = atoi(argv[1]);
    remote = atoi(argv[2]);
    sizeElement = atoi(argv[3]);

    // printf("%d\n", sizeElement);

    size_t size = sizeElement * sizeof(int);

    // Allocate input vectors in host memory
    h_local = (int*)malloc(size);
    h_remote = (int*)malloc(size);

    // Initialize input vectors, local sets to be 1, remote set to be 100
    initVec(h_local, sizeElement, 1);
    initVec(h_remote, sizeElement, 100);

    // local GPU contains d_local
    hipSetDevice(local);
    hipMalloc((void**)&d_local, size);  

    // remote GPU contains d_remote 
    hipSetDevice(remote);
    hipMalloc((void**)&d_remote, size);

    // make sure nvlink connection exists between src and det device.
    hipSetDevice(remote); // Set local device to be used for GPU executions.
    hipDeviceEnablePeerAccess(local, 0);  // Enables direct access to memory allocations on a peer device.
    hipSetDevice(local); // Set local device to be used for GPU executions.
    hipDeviceEnablePeerAccess(remote, 0);  // Enables direct access to memory allocations on a peer device.


    // Copy vector local from host memory to device memory
    hipMemcpy(d_local, h_local, size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    // Copy vector remote from host memory to device memory
    hipMemcpy(d_remote, h_remote, size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    
    int blockSize = 1;
    int gridSize = (sizeElement + blockSize - 1) / blockSize;



    hipProfilerStart();
    // Use kernel to force nvlink transaction
    for(int i = 0; i < 100; i++){


        // kernel execution 
        test_nvlink <<<blockSize, gridSize>>>(d_remote, d_local, sizeElement); 
        // hipDeviceSynchronize();


    }
    hipProfilerStop();

    // // Use hipMemcpyPeer api to launch nvlink transaction
    // for(int i = 0; i < 1000; i++){


    //     // kernel execution 
    //     hipMemcpyPeer(d_local, local, d_remote, remote, size); // copy data from remote to local
    //     hipDeviceSynchronize();


    // }


    
 


    // Copy back to host memory 
    hipMemcpy(h_remote, d_remote, size, hipMemcpyDeviceToHost); 
    hipMemcpy(h_local, d_local, size, hipMemcpyDeviceToHost); 
    hipDeviceSynchronize();
   
    double mb = sizeElement * sizeof(int) / (double)1e6;
    printf("Size of data transfer (MB): %f\n", mb);
    printf("Vector V_local (original value = 1): %d\n",h_local[sizeElement-1]);
    printf("Vector V_remote (original value = 100): %d\n",h_remote[sizeElement-1]);




    hipFree(d_local);
    hipFree(d_remote);
    free(h_local);
    free(h_remote);
    

    // exit(EXIT_SUCCESS);
 }
 