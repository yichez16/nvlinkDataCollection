/*
* Author: Yicheng Zhang
* Association: UC Riverside
* Date: Oct 3, 2023
*
* Description: 
* Description: Sender for covert-channel attack
* Idea of program: To convey bit "1", we use CUDA driver op (hipMemcpyPeer) to force memcpy from remote gpu to local gpu.
* To convey bit "0", we use std::this_thread::sleep_for(std::chrono::microseconds(1000)) to block the execution of the current thread.
* The value of d_local, d_remote vector are set to be 1 and 100.
* Four input values for this program: local gpu ID, remote gpu ID, size of data transfer and sleep time (us).
*/

#include <vector>
#include <hip/hip_runtime_api.h> // For hipProfilerStart() and hipProfilerStop()
#include <cstdio>
#include <string>
#include <thrust/device_vector.h>
#include <fstream>
#include <cupti_profiler.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include "kernel.cu"
#include <iostream>
#include <chrono>
#include <thread>




 
int main(int argc, char **argv) {

    using namespace std;
    int local = 0; 
    int remote = 1;
    int sizeElement = 64;
    int time2sleep = 1000;
    int *h_local, *h_remote;
    int *d_local, *d_remote;
    
    struct timeval ts,te, te1, te2, te3 ;

    local = atoi(argv[1]);
    remote = atoi(argv[2]);
    sizeElement = atoi(argv[3]);
    time2sleep = atoi(argv[4]);

    // printf("%d\n", sizeElement);

    size_t size = sizeElement * sizeof(int);

    // set up profiler
    hipSetDevice(local);
    hipDevice_t device;
    DRIVER_API_CALL(hipInit(0));  
	// Initialize the CUDA driver API Initializes the driver API and must be called before any other function from the driver API in the current process. Currently, the Flags parameter must be 0. If hipInit() has not been called, any function from the driver API will return hipErrorNotInitialized.
	DRIVER_API_CALL(hipDeviceGet(&device, local));
	// Returns a handle to a compute device.

    // define ctrs to profile
	vector<string> event_names {        
               
	};
	vector<string> metric_names {
    // ctr_num
	// "l2_read_transactions",// works
	//"nvlink_data_receive_efficiency",
	// "nvlink_data_transmission_efficiency",
	//"nvlink_overhead_data_received",
	//"nvlink_overhead_data_transmitted",
	//"nvlink_receive_throughput",
	"nvlink_total_data_received",// works
	//"pcie_total_data_received",
	// "nvlink_total_data_transmitted",// works
	//  "nvlink_total_nratom_data_transmitted" , // works
	// "nvlink_total_ratom_data_transmitted" ,
	//  "nvlink_total_response_data_received" ,// works
	// "nvlink_total_write_data_transmitted",
	// "nvlink_transmit_throughput", //works
	// "nvlink_user_data_received",
	// "nvlink_user_data_transmitted",
	// "nvlink_user_nratom_data_transmitted" ,
	// "nvlink_user_ratom_data_transmitted",
	// "nvlink_user_response_data_received",
	// "nvlink_user_write_data_transmitted",

	// "l2_write_transactions",  // error
	//"dram_read_transactions",
	//"dram_write_transactions",

						
	};
    hipCtx_t context;
	hipCtxCreate(&context, 0, local); // context is created on device # profile

    // Allocate input vectors in host memory
    h_local = (int*)malloc(size);
    h_remote = (int*)malloc(size);

    // Initialize input vectors, local sets to be 1, remote set to be 100
    initVec(h_local, sizeElement, 1);
    initVec(h_remote, sizeElement, 100);

    // local GPU contains d_local
    hipSetDevice(local);
    hipMalloc((void**)&d_local, size);  

    // remote GPU contains d_remote 
    hipSetDevice(remote);
    hipMalloc((void**)&d_remote, size);

    // make sure nvlink connection exists between src and det device.
    hipSetDevice(remote); // Set local device to be used for GPU executions.
    hipDeviceEnablePeerAccess(local, 0);  // Enables direct access to memory allocations on a peer device.
    hipSetDevice(local); // Set local device to be used for GPU executions.
    hipDeviceEnablePeerAccess(remote, 0);  // Enables direct access to memory allocations on a peer device.


    // Copy vector local from host memory to device memory
    hipMemcpy(d_local, h_local, size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    // Copy vector remote from host memory to device memory
    hipMemcpy(d_remote, h_remote, size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    
    int blockSize = 1;
    int gridSize = (sizeElement + blockSize - 1) / blockSize;


    std::this_thread::sleep_for(std::chrono::seconds(2));   // wait for synchronization

    // synchronization: trojan send data through nvlink and let spy know ready to send

    // for(int j = 0; j < 1; j++){
    //     cupti_profiler::profiler *p= new cupti_profiler::profiler(event_names, metric_names, context);

            
    //     p->start();
    //     gettimeofday(&ts,NULL);
    //     // hipMemcpyPeer(d_local, local, d_remote, remote, size); // copy data from remote to local

    //     test_nvlink <<<gridSize, blockSize>>>(d_remote, d_local, 100000); // 56 SMs, 4*32 =  128 threads  (src, det, numElements)  force to transfer data from remote to local.
    //     p->stop();
    //     hipDeviceSynchronize();

    //     gettimeofday(&te,NULL);
    //     // p->print_event_values(std::cout,ts,te);
    //     p->print_metric_values(std::cout,ts,te);

    //     printf("\n"); 
    //     free(p);

    // }
    

    
    gettimeofday(&ts, NULL);  
    std::cout   
    << ts.tv_sec*1000000 + ts.tv_usec
    // << ","
    // << te.tv_sec*1000000 + te.tv_usec
    ;
    // synchronization 1000 consecutive "1"
    printf("\n");

    hipDeviceSynchronize();

    for(int i = 0; i < 10000; i++){

        // Start record time

        // kernel execution
        test_nvlink <<<blockSize, gridSize>>>(d_remote, d_local, sizeElement); 
        // std::this_thread::sleep_for(std::chrono::microseconds(time2sleep)); 
        hipDeviceSynchronize();

        // test_nvlink <<<blockSize, gridSize>>>(d_remote, d_local, sizeElement); 
        // // std::this_thread::sleep_for(std::chrono::microseconds(10)); 
        // hipDeviceSynchronize();

        // std::this_thread::sleep_for(std::chrono::microseconds(1));       
        // gettimeofday(&te, NULL);  

        
        // test_nvlink <<<gridSize, blockSize>>>(d_remote, d_local, sizeElement); 
        // std::this_thread::sleep_for(std::chrono::microseconds(1));       
        // gettimeofday(&te1, NULL);  
        // hipDeviceSynchronize();



        std::this_thread::sleep_for(std::chrono::microseconds(time2sleep)); // Sleep for 1 millisecond (1000 microseconds)
        
        // std::this_thread::sleep_for(std::chrono::microseconds(time2sleep)); // Sleep for 1 millisecond (1000 microseconds)

        // gettimeofday(&te2, NULL);  

        // std::this_thread::sleep_for(std::chrono::microseconds(time2sleep)); // Sleep for 1 millisecond (1000 microseconds)

        // gettimeofday(&te3, NULL);  





        // // Print out start and stop time
        // std::cout   << size
        // << "," 
        // << (te.tv_sec - ts.tv_sec) * 1000000 + (te.tv_usec - ts.tv_usec)
        // << "," 
        // << (te1.tv_sec - te.tv_sec) * 1000000 + (te1.tv_usec - te.tv_usec)
        // << "," 
        // << (te2.tv_sec - te1.tv_sec) * 1000000 + (te2.tv_usec - te1.tv_usec)
        // << "," 
        // << (te3.tv_sec - te2.tv_sec) * 1000000 + (te3.tv_usec - te2.tv_usec)
        // ;
        // printf("\n"); 

        hipDeviceSynchronize();


    }
    gettimeofday(&te, NULL);  
    std::cout   
    << te.tv_sec*1000000 + te.tv_usec
    // << ","
    // << te.tv_sec*1000000 + te.tv_usec
    ;
    printf("\n"); 


    // // Emd of communication
    // for(int j = 0; j < 1; j++){
    //     cupti_profiler::profiler *p= new cupti_profiler::profiler(event_names, metric_names, context);

            
    //     p->start();
    //     gettimeofday(&ts,NULL);
    //     hipMemcpyPeer(d_local, local, d_remote, remote, size); // copy data from remote to local

    //     test_nvlink <<<gridSize, blockSize>>>(d_remote, d_local, 100000); // 56 SMs, 4*32 =  128 threads  (src, det, numElements)  force to transfer data from remote to local.
    //     p->stop();
    //     hipDeviceSynchronize();

    //     gettimeofday(&te,NULL);
    //     // p->print_event_values(std::cout,ts,te);
    //     p->print_metric_values(std::cout,ts,te);

    //     printf("\n"); 
    //     free(p);

    // }    


 


    // Copy back to host memory 
    hipMemcpy(h_remote, d_remote, size, hipMemcpyDeviceToHost); 
    hipMemcpy(h_local, d_local, size, hipMemcpyDeviceToHost); 
    hipDeviceSynchronize();
   
    double mb = sizeElement * sizeof(int) / (double)1e6;
    // printf("Size of data transfer (MB): %f\n", mb);
    // printf("Vector V_local (original value = 1): %d\n",h_local[sizeElement-1]);
    // printf("Vector V_remote (original value = 100): %d\n",h_remote[sizeElement-1]);




    hipFree(d_local);
    hipFree(d_remote);
    free(h_local);
    free(h_remote);
    

    // exit(EXIT_SUCCESS);
 }
 